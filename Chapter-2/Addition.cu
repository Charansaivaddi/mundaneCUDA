#include <iostream>
#include <hip/hip_runtime.h>

__global__
void vecAddKernel(float *h_A, float *h_B, float *h_C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n) h_C[i] = h_A[i] + h_B[i];
}


void vecAdd(float* A, float* B, float* C, int n)
{
    int size = n * sizeof(float);
    float *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_B, size);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_C, size);

    vecAddKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);

    hipDeviceSynchronize();

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main()
{
    float* A = new float[1000];
    float* B = new float[1000];
    float* C = new float[1000];

    int n = 1000;
    vecAdd(A, B, C, n);

}
